#include "hip/hip_runtime.h"
#include "common.h"

struct load_test {
    template<typename T>
    __host__ __device__ void operator()(generator<T> gen) {
        T data[8] = {T(0), T(1), T(2), T(3), T(4), T(5), T(6), T(7)};

        {
            auto expected = kf::make_vec(T(3), T(2), T(7));
            auto output = kf::load(data, kf::make_vec(3, 2, 7));
            ASSERT_EQ(expected, output);
        }

        {
            auto expected = kf::make_vec(T(3), T(2), T(7));
            auto output = kf::load(data, kf::make_vec(3, 2, 7), kf::make_vec(true, true, true));
            ASSERT_EQ(expected, output);
        }

        {
            auto expected = kf::make_vec(T(3), T(), T(7));
            auto output = kf::load(data, kf::make_vec(3, 100, 7), kf::make_vec(true, false, true));
            ASSERT_EQ(expected, output);
        }

        {
            auto expected = kf::make_vec(T(0), T(1), T(2));
            auto output = kf::loadn<3>(data);
            ASSERT_EQ(expected, output);
        }

        {
            auto expected = kf::make_vec(T(2), T(3), T(4));
            auto output = kf::loadn<3>(data, 2);
            ASSERT_EQ(expected, output);
        }

        {
            auto expected = kf::make_vec(T(6), T(7), T());
            auto output = kf::loadn<3>(data, 6, 8);
            ASSERT_EQ(expected, output);
        }
    }
};

REGISTER_TEST_CASE("load", load_test, int, float, double)
REGISTER_TEST_CASE_GPU("load", load_test, __half, __hip_bfloat16)

struct store_test {
    template<typename T>
    __host__ __device__ void operator()(generator<T> gen) {
        {
            T data[4] = {T(0), T(1), T(2), T(3)};
            auto values = kf::make_vec(T(100), T(200));
            auto offsets = kf::make_vec(1, 3);
            kf::store(values, data, offsets);
            ASSERT_EQ(data[0], T(0));
            ASSERT_EQ(data[1], T(100));
            ASSERT_EQ(data[2], T(2));
            ASSERT_EQ(data[3], T(200));
        }

        {
            T data[4] = {T(0), T(1), T(2), T(3)};
            auto values = kf::make_vec(T(100), T(200));
            auto offsets = kf::make_vec(1, 3);
            auto mask = kf::make_vec(true, true);
            kf::store(values, data, offsets, mask);
            ASSERT_EQ(data[0], T(0));
            ASSERT_EQ(data[1], T(100));
            ASSERT_EQ(data[2], T(2));
            ASSERT_EQ(data[3], T(200));
        }

        {
            T data[4] = {T(0), T(1), T(2), T(3)};
            auto values = kf::make_vec(T(100), T(200));
            auto offsets = kf::make_vec(1, 3);
            auto mask = kf::make_vec(true, false);
            kf::store(values, data, offsets, mask);
            ASSERT_EQ(data[0], T(0));
            ASSERT_EQ(data[1], T(100));
            ASSERT_EQ(data[2], T(2));
            ASSERT_EQ(data[3], T(3));
        }

        {
            T data[4] = {T(0), T(1), T(2), T(3)};
            auto values = kf::make_vec(T(100), T(200));
            kf::storen(values, data);
            ASSERT_EQ(data[0], T(100));
            ASSERT_EQ(data[1], T(200));
            ASSERT_EQ(data[2], T(2));
            ASSERT_EQ(data[3], T(3));
        }

        {
            T data[4] = {T(0), T(1), T(2), T(3)};
            auto values = kf::make_vec(T(100), T(200));
            kf::storen(values, data, 1);
            ASSERT_EQ(data[0], T(0));
            ASSERT_EQ(data[1], T(100));
            ASSERT_EQ(data[2], T(200));
            ASSERT_EQ(data[3], T(3));
        }

        {
            T data[4] = {T(0), T(1), T(2), T(3)};
            auto values = kf::make_vec(T(100), T(200));
            kf::storen(values, data, 1, 4);
            ASSERT_EQ(data[0], T(0));
            ASSERT_EQ(data[1], T(100));
            ASSERT_EQ(data[2], T(200));
            ASSERT_EQ(data[3], T(3));
        }

        {
            T data[4] = {T(0), T(1), T(2), T(3)};
            auto values = kf::make_vec(T(100), T(200));
            kf::storen(values, data, 3, 4);
            ASSERT_EQ(data[0], T(0));
            ASSERT_EQ(data[1], T(1));
            ASSERT_EQ(data[2], T(2));
            ASSERT_EQ(data[3], T(100));
        }
    }
};

REGISTER_TEST_CASE("store", store_test, int, float, double)
REGISTER_TEST_CASE_GPU("store", store_test, __half, __hip_bfloat16)