#include "hip/hip_runtime.h"
#include "common.h"

struct load_test {
    template<typename T>
    __host__ __device__ void operator()(generator<T> gen) {
        T data[8] = {T(0.0), T(1.0), T(2.0), T(3.0), T(4.0), T(5.0), T(6.0), T(7.0)};

        {
            auto expected = kf::make_vec(T(3.0), T(2.0), T(7.0));
            auto output = kf::read(data, kf::make_vec(3, 2, 7));
            ASSERT_EQ(expected, output);
        }

        {
            auto expected = kf::make_vec(T(3.0), T(2.0), T(7.0));
            auto output = kf::read(data, kf::make_vec(3, 2, 7), kf::make_vec(true, true, true));
            ASSERT_EQ(expected, output);
        }

        {
            auto expected = kf::make_vec(T(3.0), T(), T(7.0));
            auto output = kf::read(data, kf::make_vec(3, 100, 7), kf::make_vec(true, false, true));
            ASSERT_EQ(expected, output);
        }

        {
            auto expected = kf::make_vec(T(0.0), T(1.0), T(2.0));
            auto output = kf::read<3>(data);
            ASSERT_EQ(expected, output);
        }

        {
            auto expected = kf::make_vec(T(2.0), T(3.0), T(4.0));
            auto output = kf::read<3>(data + 2);
            ASSERT_EQ(expected, output);
        }
    }
};

REGISTER_TEST_CASE("load", load_test, int, float, double, __half, __hip_bfloat16)

struct store_test {
    template<typename T>
    __host__ __device__ void operator()(generator<T> gen) {
        {
            T data[4] = {T(0.0), T(1.0), T(2.0), T(3.0)};
            auto values = kf::make_vec(T(100.0), T(200.0));
            auto offsets = kf::make_vec(1, 3);
            kf::write(data, offsets, values);
            ASSERT_EQ(data[0], T(0.0));
            ASSERT_EQ(data[1], T(100.0));
            ASSERT_EQ(data[2], T(2.0));
            ASSERT_EQ(data[3], T(200.0));
        }

        {
            T data[4] = {T(0.0), T(1.0), T(2.0), T(3.0)};
            auto values = kf::make_vec(T(100.0), T(200.0));
            auto offsets = kf::make_vec(1, 3);
            auto mask = kf::make_vec(true, true);
            kf::write(data, offsets, values, mask);
            ASSERT_EQ(data[0], T(0.0));
            ASSERT_EQ(data[1], T(100.0));
            ASSERT_EQ(data[2], T(2.0));
            ASSERT_EQ(data[3], T(200.0));
        }

        {
            T data[4] = {T(0.0), T(1.0), T(2.0), T(3.0)};
            auto values = kf::make_vec(T(100.0), T(200.0));
            auto offsets = kf::make_vec(1, 3);
            auto mask = kf::make_vec(true, false);
            kf::write(data, offsets, values, mask);
            ASSERT_EQ(data[0], T(0.0));
            ASSERT_EQ(data[1], T(100.0));
            ASSERT_EQ(data[2], T(2.0));
            ASSERT_EQ(data[3], T(3.0));
        }

        {
            T data[4] = {T(0.0), T(1.0), T(2.0), T(3.0)};
            auto values = kf::make_vec(T(100.0), T(200.0));
            kf::write(data, values);
            ASSERT_EQ(data[0], T(100.0));
            ASSERT_EQ(data[1], T(200.0));
            ASSERT_EQ(data[2], T(2.0));
            ASSERT_EQ(data[3], T(3.0));
        }

        {
            T data[4] = {T(0.0), T(1.0), T(2.0), T(3.0)};
            auto values = kf::make_vec(T(100.0), T(200.0));
            kf::write(data + 1, values);
            ASSERT_EQ(data[0], T(0.0));
            ASSERT_EQ(data[1], T(100.0));
            ASSERT_EQ(data[2], T(200.0));
            ASSERT_EQ(data[3], T(3.0));
        }
    }
};

REGISTER_TEST_CASE("store", store_test, int, float, double, __half, __hip_bfloat16)

struct assign_conversion_test {
    template<typename T, size_t... I, size_t N = sizeof...(I)>
    __host__ __device__ void operator()(generator<T> gen, std::index_sequence<I...>) {
        kf::vec<T, N> x = {gen.next(I)...};
        kf::vec<float, N> y;

        kf::cast_to(y) = x;

        ASSERT_EQ_ALL(float(x[I]), y[I]);
    }
};

REGISTER_TEST_CASE(
    "assign conversion",
    assign_conversion_test,
    int,
    float,
    double,
    __half,
    __hip_bfloat16)

struct aligned_ptr_test {
    template<typename T, size_t... I, size_t N = sizeof...(I)>
    __host__ __device__ void operator()(generator<T>, std::index_sequence<I...>) {
        struct alignas(32) storage_type {
            T data[N];
        };

        storage_type input = {T(double(I))...};
        auto v = kf::read_aligned<N>(input.data);
        ASSERT_EQ_ALL(v[I], T(double(I)));

        storage_type output = {T(double(I * 0))...};
        kf::write_aligned<N>(output.data, v);
        ASSERT_EQ_ALL(output.data[I], T(double(I)));
    }
};

REGISTER_TEST_CASE("aligned pointer", aligned_ptr_test, int, float, double, __half, __hip_bfloat16)
