#include "hip/hip_runtime.h"
#include "common.h"
#include "kernel_float.h"

namespace kf = kernel_float;

template<typename T, size_t N, typename Is = std::make_index_sequence<N>>
struct int_test;

template<typename T, size_t N, size_t... Is>
struct int_test<T, N, std::index_sequence<Is...>> {
    __host__ __device__ void operator()(generator<T> gen) {
        kf::vec<T, N> a {gen.next(Is)...};
        kf::vec<T, N> b;

        b = -a;
        ASSERT((b.get(Is) == -(a.get(Is))) && ...);

        b = ~a;
        ASSERT((b.get(Is) == ~(a.get(Is))) && ...);

        b = !a;
        ASSERT((b.get(Is) == !(a.get(Is))) && ...);
    }
};

template<typename T, size_t N, typename Is = std::make_index_sequence<N>>
struct float_test;

template<typename T, size_t N, size_t... Is>
struct float_test<T, N, std::index_sequence<Is...>> {
    __host__ __device__ void operator()(generator<T> gen) {
        kf::vec<T, N> a {gen.next(Is)...};
        kf::vec<T, N> b;

        b = -a;
        ASSERT(equals(-a.get(Is), b.get(Is)) && ...);

        // just some examples
        b = kf::cos(a);
        ASSERT(equals(cos(a.get(Is)), b.get(Is)) && ...);

        b = kf::floor(a);
        ASSERT(equals(floor(a.get(Is)), b.get(Is)) && ...);

        b = kf::abs(a);
        ASSERT(equals(abs(a.get(Is)), b.get(Is)) && ...);

        b = kf::sqrt(a);
        ASSERT(equals(sqrt(a.get(Is)), b.get(Is)) && ...);
    }
};

template<size_t N, size_t... Is>
struct float_test<__half, N, std::index_sequence<Is...>> {
    template<typename T>
    __host__ __device__ void operator()(generator<T> gen) {
        kf::vec<T, N> a {gen.next(Is)...};
        kf::vec<T, N> b;

        b = -a;
        ASSERT(equals(__hneg(a.get(Is)), b.get(Is)) && ...);

        // just some examples
        b = kf::cos(a);
        ASSERT(equals(hcos(a.get(Is)), b.get(Is)) && ...);

        b = kf::floor(a);
        ASSERT(equals(hfloor(a.get(Is)), b.get(Is)) && ...);

        b = kf::abs(a);
        ASSERT(equals(__habs(a.get(Is)), b.get(Is)) && ...);

        b = kf::sqrt(a);
        ASSERT(equals(hsqrt(a.get(Is)), b.get(Is)) && ...);
    }
};

template<size_t N, size_t... Is>
struct float_test<__hip_bfloat16, N, std::index_sequence<Is...>> {
    __host__ __device__ void operator()(generator<__hip_bfloat16> gen) {
        float_test<__half, N> {}(gen);
    }
};

TEST_CASE("unary operators") {
    run_on_host_and_device<int_test, char, short, int, unsigned, int, long, long long>();

    run_on_host_and_device<float_test, float, double>();
    run_on_device<float_test, __half, __hip_bfloat16>();
}
