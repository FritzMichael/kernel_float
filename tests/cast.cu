#include "hip/hip_runtime.h"
#include "common.h"
#include "kernel_float.h"

namespace kf = kernel_float;

template<typename A, typename B, size_t N, typename Is = std::make_index_sequence<N>>
struct cast_test;

template<typename A, typename B, size_t N, size_t... Is>
struct cast_test<A, B, N, std::index_sequence<Is...>> {
    __host__ __device__ void operator()(generator<A> gen) {
        kf::vec<A, N> a {gen.next(Is)...};
        kf::vec<B, N> b = kf::cast<B>(a);

        ASSERT(bitwise_equal(B(a.get(Is)), b.get(Is)) && ...);
    }
};

template<size_t N, size_t... Is>
struct cast_test<__half, long, N, std::index_sequence<Is...>> {
    __host__ __device__ void operator()(generator<__half> gen) {
        kf::vec<__half, N> a {gen.next(Is)...};
        kf::vec<long, N> b = kf::cast<long>(a);
        ASSERT(bitwise_equal((long)(long long)a.get(Is), b.get(Is)) && ...);
    }
};

template<size_t N, size_t... Is>
struct cast_test<long, __half, N, std::index_sequence<Is...>> {
    __host__ __device__ void operator()(generator<long> gen) {
        kf::vec<long, N> a {gen.next(Is)...};
        kf::vec<__half, N> b = kf::cast<__half>(a);
        ASSERT(bitwise_equal(__half((long long)a.get(Is)), b.get(Is)) && ...);
    }
};

template<size_t N, size_t... Is>
struct cast_test<unsigned long, __half, N, std::index_sequence<Is...>> {
    __host__ __device__ void operator()(generator<unsigned long> gen) {
        kf::vec<unsigned long, N> a {gen.next(Is)...};
        kf::vec<__half, N> b = kf::cast<__half>(a);
        ASSERT(bitwise_equal((__half)(unsigned long long)(a.get(Is)), b.get(Is)) && ...);
    }
};

template<size_t N, size_t... Is>
struct cast_test<__half, char, N, std::index_sequence<Is...>> {
    __host__ __device__ void operator()(generator<__half> gen) {
        kf::vec<__half, N> a {gen.next(Is)...};
        kf::vec<char, N> b = kf::cast<char>(a);
        ASSERT(bitwise_equal((char)(int)(a.get(Is)), b.get(Is)) && ...);
    }
};

template<size_t N, size_t... Is>
struct cast_test<__hip_bfloat16, long, N, std::index_sequence<Is...>> {
    __host__ __device__ void operator()(generator<__hip_bfloat16> gen) {
        kf::vec<__hip_bfloat16, N> a {gen.next(Is)...};
        kf::vec<long, N> b = kf::cast<long>(a);
        ASSERT(bitwise_equal((long)(long long)a.get(Is), b.get(Is)) && ...);
    }
};

template<size_t N, size_t... Is>
struct cast_test<long, __hip_bfloat16, N, std::index_sequence<Is...>> {
    __host__ __device__ void operator()(generator<long> gen) {
        kf::vec<long, N> a {gen.next(Is)...};
        kf::vec<__hip_bfloat16, N> b = kf::cast<__hip_bfloat16>(a);
        ASSERT(bitwise_equal(__hip_bfloat16((long long)a.get(Is)), b.get(Is)) && ...);
    }
};

template<size_t N, size_t... Is>
struct cast_test<unsigned long, __hip_bfloat16, N, std::index_sequence<Is...>> {
    __host__ __device__ void operator()(generator<unsigned long> gen) {
        kf::vec<unsigned long, N> a {gen.next(Is)...};
        kf::vec<__hip_bfloat16, N> b = kf::cast<__hip_bfloat16>(a);
        ASSERT(bitwise_equal((__hip_bfloat16)(unsigned long long)(a.get(Is)), b.get(Is)) && ...);
    }
};

template<size_t N, size_t... Is>
struct cast_test<__hip_bfloat16, char, N, std::index_sequence<Is...>> {
    __host__ __device__ void operator()(generator<__hip_bfloat16> gen) {
        kf::vec<__hip_bfloat16, N> a {gen.next(Is)...};
        kf::vec<char, N> b = kf::cast<char>(a);
        ASSERT(bitwise_equal((char)(int)(a.get(Is)), b.get(Is)) && ...);
    }
};

template<size_t N, size_t... Is>
struct cast_test<__hip_bfloat16, __half, N, std::index_sequence<Is...>> {
    __host__ __device__ void operator()(generator<__hip_bfloat16> gen) {
        kf::vec<__hip_bfloat16, N> a {gen.next(Is)...};
        kf::vec<__half, N> b = kf::cast<__half>(a);
        ASSERT(bitwise_equal((__half)(float)(a.get(Is)), b.get(Is)) && ...);
    }
};

template<size_t N, size_t... Is>
struct cast_test<__half, __hip_bfloat16, N, std::index_sequence<Is...>> {
    __host__ __device__ void operator()(generator<__half> gen) {
        kf::vec<__half, N> a {gen.next(Is)...};
        kf::vec<__hip_bfloat16, N> b = kf::cast<__hip_bfloat16>(a);
        ASSERT(bitwise_equal((__hip_bfloat16)(float)(a.get(Is)), b.get(Is)) && ...);
    }
};

template<typename B>
struct cast_to {
    template<typename A, size_t N>
    using type = cast_test<A, B, N>;
};

TEST_CASE("cast operators") {
    auto types = type_sequence<
        bool,
        char,
        short,
        int,
        unsigned int,
        long,
        unsigned long,
        long long,
        float,
        double,
        __half,
        __hip_bfloat16> {};

    run_on_host_and_device<cast_to<bool>::template type>(types);
    run_on_host_and_device<cast_to<char>::template type>(types);
    run_on_host_and_device<cast_to<short>::template type>(types);
    run_on_host_and_device<cast_to<int>::template type>(types);
    run_on_host_and_device<cast_to<long>::template type>(types);
    run_on_host_and_device<cast_to<long long>::template type>(types);
    run_on_host_and_device<cast_to<__half>::template type>(types);
    run_on_host_and_device<cast_to<__hip_bfloat16>::template type>(types);
    run_on_host_and_device<cast_to<float>::template type>(types);
    run_on_host_and_device<cast_to<double>::template type>(types);

    //bool, char, short, int, long long, __half, float, double
}
