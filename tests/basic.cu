#include "hip/hip_runtime.h"
#include "catch.hpp"
#include "common.h"
#include "kernel_float.h"

namespace kf = kernel_float;

template<typename T, size_t N, typename Is = std::make_index_sequence<N>>
struct basic_test;

template<typename T, size_t N, size_t... Is>
struct basic_test<T, N, std::index_sequence<Is...>> {
    __host__ __device__ void operator()(generator<T> gen) {
        T items[N] = {gen.next(Is)...};
        kf::vec<T, N> a = {items[Is]...};

        // check if getters work
        ASSERT(bitwise_equal(a.get(Is), items[Is]) && ...);
        ASSERT(bitwise_equal(a.get(kf::constant_index<Is> {}), items[Is]) && ...);
        ASSERT(bitwise_equal<T>(a[Is], items[Is]) && ...);
        ASSERT(bitwise_equal<T>(a[kf::constant_index<Is> {}], items[Is]) && ...);

        // check if setter works
        T new_items[N] = {gen.next(Is)...};
        (a.set(kf::constant_index<Is> {}, new_items[Is]), ...);
        ASSERT(bitwise_equal(a.get(Is), new_items[Is]) && ...);

        // check if setter works
        T more_new_items[N] = {gen.next(Is)...};
        ((a[Is] = more_new_items[Is]), ...);
        ASSERT(bitwise_equal(a.get(Is), more_new_items[Is]) && ...);

        // check default constructor
        kf::vec<T, N> b;
        ASSERT(bitwise_equal(b.get(Is), T {}) && ...);

        // check broadcast constructor
        T value = gen();
        kf::vec<T, N> c {value};
        ASSERT(bitwise_equal(c.get(Is), value) && ...);

        // check make_vec
        kf::vec<T, N> d = kf::make_vec(items[Is]...);
        ASSERT(bitwise_equal(d.get(Is), items[Is]) && ...);
    }
};

TEST_CASE("basic") {
    run_on_host_and_device<basic_test, bool, int, float, double>();
    run_on_device<basic_test, __half>();
}